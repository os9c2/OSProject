#include "hip/hip_runtime.h"
/*********************************************
 *  knnfunc.c
 *  Sample KNN implementation
 *  COMP9417 Machine Learning and Data Mining
 *  UNSW Session 1, 2012
 *	Chengbin Wang z3313137

 *  KNN functions implementation
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <string.h>
#include "knn.cuh"


//struct Node SampleCopy[MAX_SAMPLE];
float square(float t){
	return t*t;
}

float absv(float a, float b){
	if (a > b) return a-b;
	return b-a;
}

// CALCULATE DISTANCE OF EACH NODE
__global__ void getdistances(float *device_array,int predict_attr,struct Node* Sample, float* actual,int *Sample_Num,int *Atttr_Num,char *Atttr_Seq){
	float dist;
	int i,j,k;
	for(i = 0; i < *Sample_Num; i++){
		for(j = 0, dist = 0, k = 0; j < *Atttr_Num; j++, k++){
			//use the prediction attribute
			if(j == predict_attr){
				
			// if it is an real number, use ecludiean distance
			if(Atttr_Seq[k] == 'r'){
				dist += (device_array[i*(*Atttr_Num)+j] - actual[j])*(device_array[i*(*Atttr_Num)+j] - actual[j]);
			}
		    }
		}
		Sample[i].distance = sqrt(dist);
	}
}

// quick sort the Nodes with distances
void quicksort(int low, int high,struct Node* Sample){

	int i,j,piviot;
	struct Node t;

	if(high > low){
		piviot = (low+high)/2;
		SWAP(t,Sample[low],Sample[piviot]);

		i = low + 1;
		j = high;
		while(i <= j){
			while(Sample[i].distance <= Sample[low].distance
				  && i <= high){i++;}
			while(Sample[j].distance >= Sample[low].distance
				  && j > low){j--;}
			if(i < j){
				SWAP(t,Sample[i],Sample[j]);
			}
		}
		SWAP(t,Sample[low],Sample[j]);
		quicksort ( low, j - 1,Sample );
		quicksort ( j + 1, high,Sample );
	}
}


//KNN to predict numeric value
float NUM_KNN(int index,int k,int predict_attr,struct Node* Sample,int *Sample_Num,int *Atttr_Num,char *Atttr_Seq,int predict_attr_count,float *actual){
	int i,j;
	float result = 0,sum=0;
	int num_bytes=(*Sample_Num) * (*Atttr_Num) * sizeof(float);
	float *device_array = 0;
  	float *host_array = 0;
	host_array = (float*)malloc(num_bytes);	
	for(i=0;i<*Sample_Num;i++)
		for(j=0;j<*Atttr_Num;j++)
			host_array[i*(*Atttr_Num)+j]=Sample[i].NumericAttributes[j];
	hipMalloc((void**)&device_array, num_bytes);
	hipMemcpy(device_array, host_array, num_bytes, hipMemcpyHostToDevice);
	dim3 block_size;
 	block_size.x = *Sample_Num;
  	block_size.y = 1;

  	// configure a two dimensional grid as well
  	dim3 grid_size;
  	grid_size.x = *Sample_Num / block_size.x;
  	grid_size.y = *Atttr_Num / block_size.y;
	
	getdistances<<<grid_size,block_size>>>(device_array,predict_attr,Sample,actual,Sample_Num,Atttr_Num,Atttr_Seq);	
	hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

	free(host_array);   
  	hipFree(device_array);

	quicksort(0,*Sample_Num-1,Sample);

	
	// vote and get average
	for(i = 1; i <= k ; i++){
		result += Sample[i].NumericAttributes[predict_attr];
	}
		result = result/k;
	return result; 

}

